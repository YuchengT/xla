#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>

#include <iostream>

#include "custom.h"
#include "tensorflow/compiler/xla/service/custom_call_target_registry.h"
#include "tensorflow/compiler/xla/xla_data.pb.h"

__global__ void saxpy(int n, float a, float* x, float* y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

void test() {
  int N = 1 << 20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N * sizeof(float));
  y = (float*)malloc(N * sizeof(float));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = max(maxError, abs(y[i] - 4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

__global__ void custom_call_kernel(const float* input, const int64_t* index,
                                   float* out, const int64_t len,
                                   const int64_t dim) {
  const int n = 10;
  for (size_t thread_id = n * threadIdx.x; thread_id + n < len;
       thread_id += n * blockDim.x) {
    for (int k = 0; k < n; k++) {
      int64_t idx = index[thread_id + k];
      for (size_t i = 0; i < dim; i++) {
        out[(thread_id + k) * dim + i] = input[idx * dim + i];
      }
    }
  }
}

void do_custom_call(hipStream_t stream, void** buffers, const char* opaque,
                    size_t opaque_len) {
  auto start = std::chrono::steady_clock::now();
  const float* input = reinterpret_cast<const float*>(buffers[0]);
  const int64_t* index = reinterpret_cast<const int64_t*>(buffers[1]);
  float* output = reinterpret_cast<float*>(buffers[2]);
  xla::ShapeProto shape;
  shape.ParseFromArray(opaque, opaque_len);
  const int64_t len = shape.dimensions(0);
  const int64_t dim = shape.dimensions(1);
  const int64_t block_dim = 1024;
  const int64_t grid_dim = 1;
  // custom_call_kernel<<<grid_dim, block_dim,
  //                      /*dynamic_shared_mem_bytes=*/0, stream>>>(input,
  //                      index, output, len, dim);
  auto end = std::chrono::steady_clock::now();
  std::cout << "dt="
            << std::chrono::duration_cast<std::chrono::microseconds>(end -
                                                                     start)
                   .count()
            << " us" << std::endl;
}
XLA_REGISTER_CUSTOM_CALL_TARGET(do_custom_call, "CUDA");

static hiprandGenerator_t gen;
static std::once_flag cuda_rng;

void do_custom_rand(hipStream_t stream, void** buffers, const char* opaque,
                    size_t opaque_len) {
  // auto start = std::chrono::steady_clock::now();
  float* output = reinterpret_cast<float*>(buffers[0]);
  xla::ShapeProto shape;
  shape.ParseFromArray(opaque, opaque_len);
  int64_t len = 1;
  for (int i = 0; i < shape.dimensions().size(); i++) {
    len *= shape.dimensions(i);
  }
  // std::cout << "len=" << len << std::endl;
  std::call_once(cuda_rng, []() {
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  });
  hiprandSetStream(gen, stream);
  hiprandGenerateUniform(gen, output, len);
  // auto end = std::chrono::steady_clock::now();
  // std::cout << "dt="
  //           << std::chrono::duration_cast<std::chrono::microseconds>(end -
  //                                                                    start)
  //                  .count()
  //           << " us" << std::endl;
}
XLA_REGISTER_CUSTOM_CALL_TARGET(do_custom_rand, "CUDA");

__global__ void set_value(float* output, const int64_t len) {
  for (size_t thread_id = threadIdx.x; thread_id < len;
       thread_id += blockDim.x) {
    output[thread_id] = 0.5;
  }
}

void do_custom_rand2(hipStream_t stream, void** buffers, const char* opaque,
                     size_t opaque_len) {
  auto start = std::chrono::steady_clock::now();
  float* output = reinterpret_cast<float*>(buffers[0]);
  xla::ShapeProto shape;
  shape.ParseFromArray(opaque, opaque_len);
  int64_t len = 1;
  for (int i = 0; i < shape.dimensions().size(); i++) {
    len *= shape.dimensions(i);
  }
  std::cout << "len=" << len << std::endl;
  // const int64_t block_dim = 1024;
  // const int64_t grid_dim = 1;
  // set_value<<<grid_dim, block_dim,
  //             /*dynamic_shared_mem_bytes=*/0, stream>>>(output, len);
  std::call_once(cuda_rng, []() {
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  });
  hiprandGenerateUniform(gen, output, len);
  auto end = std::chrono::steady_clock::now();
  std::cout << "dt="
            << std::chrono::duration_cast<std::chrono::microseconds>(end -
                                                                     start)
                   .count()
            << " us" << std::endl;
  float* tmp = (float*)malloc(len * sizeof(float));
  hipMemcpy(tmp, output, len * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < len; i++) {
    printf("%1.4f ", tmp[i]);
  }
  printf("\n");
}
XLA_REGISTER_CUSTOM_CALL_TARGET(do_custom_rand2, "CUDA");
